#include "hip/hip_runtime.h"
//#define __device__
//#define __global__

#include "hip/hip_runtime.h"
#include ""
#include "cuda_string.cuh"
#include "cuda_list.cuh"
//#include "cuda_stack.cuh"
#include <stdio.h>
#include "CudaParser.cuh"

__device__ cu::Mathexpr<double>* g_pExpr;

__global__ void memset_expr(double* vec, std::size_t n, const char* pStr, std::size_t cbStr)
{
	auto i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i == 0)
		g_pExpr = new cu::Mathexpr<double>(pStr, cbStr);
	__syncthreads();
	if (i < n)
	{
		auto& m = *g_pExpr;
		vec[i] = m(i).value();
	}
	__syncthreads();
	if (!i)
		delete g_pExpr;
}

int main()
{
	hipError_t cudaStatus;
	const char pStr[] = "f(x) = x + 1";
	//const char pStr[] = "f(x) = 2*yn(1,0.1*3.14*sin(x)) / (0.1*3.14*sin(x))";
	double V[1000];
	std::size_t cbStack;

	cudaStatus = hipDeviceGetLimit(&cbStack, hipLimitStackSize);
	if (cudaStatus != 0)
		return -6;

	cudaStatus = hipDeviceSetLimit(hipLimitStackSize, 1 << 13);
	if (cudaStatus != 0)
		return -5;

	auto pStr_d = make_cuda_unique_ptr<char>(sizeof(pStr));
	auto V_d = make_cuda_unique_ptr<double>(sizeof(V) / sizeof(double));

	cudaStatus = hipMemcpy(pStr_d.get(), pStr, sizeof(pStr) - 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -1;
	memset_expr<<<1, sizeof(V) / sizeof(double)>>>(V_d.get(), sizeof(V) / sizeof(double), pStr_d.get(), sizeof(pStr) - 1);

	/*cuda_string expression = "f(x, y) = min(x, 5, y) + min(y, 5, x) + max(x, 5, y) + max(y, 5, x)";
	Mathexpr<double> mathexpr(expression);
	cuda_vector<double> v;
	v.push_back(1);
	v.push_back(10);
	mathexpr.init_variables(v);*/
	//std::cout << "Value: " << mathexpr.compute() << "\n";

	//cuda_list<double> l;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		return -2;
	}

	cudaStatus = hipMemcpy(V, V_d.get(), sizeof(V), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -3;

	//printf("%d", l.front());

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -4;
	}

	for (auto elem:V)
		std::cout << elem << " ";
	std::cout << "\n";

	return 0;
}
