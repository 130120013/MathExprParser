#include "hip/hip_runtime.h"
//#define __device__
//#define __global__

#include "hip/hip_runtime.h"
#include ""
#include "cuda_config.cuh"
#include "cuda_string.cuh"
#include "cuda_list.cuh"
#include "cuda_return_wrapper.cuh"
//#include "cuda_stack.cuh"
#include <stdio.h>
#include "CudaParser.cuh"

typedef thrust::complex<double> number_type;

__device__ cu::Mathexpr<number_type>* g_pExpr;

__global__ void memset_expr(cu::CudaParserErrorCodes* pCode, number_type* vec, std::size_t n, const char* pStr, std::size_t cbStr)
{
	auto i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i == 0)
		g_pExpr = new cu::Mathexpr<number_type>(pCode, pStr, cbStr);
	__syncthreads();
	if (*pCode != cu::CudaParserErrorCodes::Success)
		return;
	if (i < n)
	{
		auto& m = *g_pExpr;
		auto rv = m(number_type(i, i));
		*pCode = rv.return_code();
		if (bool(rv))
			vec[i] = rv.value();
	}
	__syncthreads();
	if (!i)
		delete g_pExpr;
}

int main()
{
	hipError_t cudaStatus;
	//const char pStr[] = "f(x) = 2*j1(0.1*3.14*sin(x)) / (0.1*3.14*sin(x))";
	const char pStr[] = "f(x) = abs(x) * (cos(arg(x)) + i * sin(arg(x)))";
	number_type V[1];
	std::size_t cbStack;

	cudaStatus = hipDeviceGetLimit(&cbStack, hipLimitStackSize);
	if (cudaStatus != 0)
		return -6;

	cudaStatus = hipDeviceSetLimit(hipLimitStackSize, 1 << 13);
	if (cudaStatus != 0)
		return -5;

	auto pStr_d = make_cuda_unique_ptr<char>(sizeof(pStr));
	if (!pStr_d)
		return -100;
	auto V_d = make_cuda_unique_ptr<number_type>(sizeof(V) / sizeof(number_type));
	if (!V_d)
		return -100;
	auto pCode = make_cuda_unique_ptr<cu::CudaParserErrorCodes>();
	if (!pCode)
		return -100;

	cudaStatus = hipMemcpy(pStr_d.get(), pStr, sizeof(pStr) - 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -1;
	memset_expr<<<1, sizeof(V) / sizeof(number_type)>>>(pCode.get(), V_d.get(), sizeof(V) / sizeof(number_type), pStr_d.get(), sizeof(pStr) - 1);

	/*cuda_string expression = "f(x, y) = min(x, 5, y) + min(y, 5, x) + max(x, 5, y) + max(y, 5, x)";
	Mathexpr<double> mathexpr(expression);
	cuda_vector<double> v;
	v.push_back(1);
	v.push_back(10);
	mathexpr.init_variables(v);*/
	//std::cout << "Value: " << mathexpr.compute() << "\n";

	//cuda_list<double> l;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		return -2;
	}

	cu::CudaParserErrorCodes errc;
	cudaStatus = hipMemcpy(&errc, pCode.get(), sizeof(cu::CudaParserErrorCodes), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -3;
	if (errc == cu::CudaParserErrorCodes::Success)
	{
		cudaStatus = hipMemcpy(V, V_d.get(), sizeof(V), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
			return -3;
		for (auto elem:V)
			std::cout << elem << " ";
		std::cout << "\n";
	}else
	{
		printf("CUDA kernel returned code %d (%s)", int(errc), strerror(errc));
		return -50;
	}

	//printf("%d", l.front());

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -4;
	}

	

	return 0;
}
