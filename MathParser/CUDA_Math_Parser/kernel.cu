#include "hip/hip_runtime.h"
//#define __device__
//#define __global__

#include "hip/hip_runtime.h"
#include ""
#include "cuda_string.cuh"
//#include "cuda_list.cuh"
//#include "cuda_stack.cuh"
#include <stdio.h>
//#include "CudaParser.h"

__global__ void memset_expr(double* vec, std::size_t n, const char* pStr, std::size_t cbStr)
{
	auto i = threadIdx.x * blockIdx.x * blockDim.x;
	if (i < n)
		vec[i] = cu::stod(cu::cuda_string(pStr, pStr + cbStr));
}

int main()
{
	hipError_t cudaStatus;
	const char pStr[] = "3.14";
	double V[1000];

	auto pStr_d = make_cuda_unique_ptr<char>(sizeof(pStr));
	auto V_d = make_cuda_unique_ptr<double>(sizeof(V) / sizeof(double));

	cudaStatus = hipMemcpy(pStr_d.get(), pStr, sizeof(pStr) - 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -1;
	memset_expr<<<2, 500>>>(V_d.get(), sizeof(V) / sizeof(double), pStr_d.get(), sizeof(pStr) - 1);

	/*cuda_string expression = "f(x, y) = min(x, 5, y) + min(y, 5, x) + max(x, 5, y) + max(y, 5, x)";
	Mathexpr<double> mathexpr(expression);
	cuda_vector<double> v;
	v.push_back(1);
	v.push_back(10);
	mathexpr.init_variables(v);*/
	//std::cout << "Value: " << mathexpr.compute() << "\n";

	//cuda_list<double> l;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		return -2;
	}

	cudaStatus = hipMemcpy(V, V_d.get(), sizeof(V), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -3;

	//printf("%d", l.front());

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -4;
	}

	return 0;
}
